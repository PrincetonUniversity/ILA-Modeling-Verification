#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;
#define imin(a,b) (a<b?a:b)

const int N = 4;
const int threadsPerBlock = 4;
const int blocksPerGrid = 1;

__global__ void dot(int *a) {
//int gid = threadIdx.x + blockIdx.x * blockDim.x; 
//int i = a[gid];
//int j = b[gid];
//int k = i + j;
    int tid = threadIdx.x;
    if (tid > 1){
        //for (int i = 0 ;i < 10; i++){
        //a[N+i] = a[tid];
        //}
        int t0 = a[tid - 2];
        int t1 = a[tid];
        //__syncthreads();
        a[tid] = t0+t1;

    } else {
        int t0 = a[tid];
        int t1 = a[tid + 2];
        //__syncthreads();
        a[tid] = t0 + t1;
 
    }
//    else
//        c[gid] =  1;
//    c[gid] = c[gid] + 1;
}
//__global__ void dot2(int *a, int *b, int*c) {
//int gid = threadIdx.x + blockIdx.x * blockDim.x;
//int i = a[gid];
//int j = b[gid];
//int k = i + j;
//}
//__global__ void mykernel(int *data){
//    atomicAdd(data, 10); 
//}
   
int main(){
    int *a;
    int *dev_a;
    
    a = new int[N];
    for (int i = 0; i < N; i++) {
        a[i] = i;
    }
    hipMalloc((void **)&dev_a, sizeof(int) * N);
    hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
    dot<<<blocksPerGrid, threadsPerBlock>>>(dev_a);
    hipMemcpy( a, dev_a, N*sizeof(int), hipMemcpyDeviceToHost);
    #define sum_sq(x) (x*(x+1)*(2*x+1)/6)

    for (int i = 0;i < N; i++)
        printf("%d\n", a[i]);
    hipFree(dev_a);
    

    delete[]  a;
}
