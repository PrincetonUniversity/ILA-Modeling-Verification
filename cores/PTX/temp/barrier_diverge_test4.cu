#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;
#define imin(a,b) (a<b?a:b)

const int N = 128;
const int threadsPerBlock = 128;
const int blocksPerGrid = 1;

__global__ void dot(int *a) {
//int gid = threadIdx.x + blockIdx.x * blockDim.x; 
//int i = a[gid];
//int j = b[gid];
//int k = i + j;
    int tid = threadIdx.x;
    for (int i = 0; i < 1; i ++) {
    if (tid >= N/2){
        int t0=0;
        int t1=0;
        for ( int j = 0 ;j < tid; j++) {
            a[N + j] = a[tid];
        }
        t0 = a[tid - N/2];
        t1 = a[tid];
        __syncthreads();
        a[tid] = t0+t1;
        __syncthreads();
        //__syncthreads();

    } else {
        int t0 = a[tid];
        int t1 = a[tid + N/2];
        //__syncthreads();
        a[tid] = t0 + t1;
        __syncthreads();
 
    }
    __syncthreads();
    }
//    else
//        c[gid] =  1;
//    c[gid] = c[gid] + 1;
}
//__global__ void dot2(int *a, int *b, int*c) {
//int gid = threadIdx.x + blockIdx.x * blockDim.x;
//int i = a[gid];
//int j = b[gid];
//int k = i + j;
//}
//__global__ void mykernel(int *data){
//    atomicAdd(data, 10); 
//}
   
int main(){
    int *a;
    int *dev_a;
    
    a = new int[N];
    for (int i = 0; i < N; i++) {
        a[i] = i;
    }
    hipMalloc((void **)&dev_a, sizeof(int) * N);
    hipMemcpy(dev_a, a, N*sizeof(int), hipMemcpyHostToDevice);
    dot<<<blocksPerGrid, threadsPerBlock>>>(dev_a);
    hipMemcpy( a, dev_a, N*sizeof(int), hipMemcpyDeviceToHost);
    #define sum_sq(x) (x*(x+1)*(2*x+1)/6)

    for (int i = 0;i < N/2; i++)
        printf("%d  %d\n", a[i], a[i + N/2]);
    hipFree(dev_a);
    

    delete[]  a;
}
